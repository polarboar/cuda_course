#include "hip/hip_runtime.h"
#include <iostream>
using namespace std;

__global__ 
void VecAdd(int* A, int* B, int* C) {
    int i = threadIdx.x;
    printf("Thread Id: %d\n", i);
    C[i] = A[i] + B[i];
}

void printVector(int* A, int len) {
    for(int i=0; i<len; i++) {
        cout << A[i] << ' ';
    }
    cout << endl;
}

int main() {
    int N = 4;

    int A[N] = {1,2,3,4};
    int B[N] = {1,2,3,4};
    int C[N] = {0,0,0,0};

    // Initialize data on device
    int *A_d, *B_d, *C_d;
    hipMalloc((void **)&A_d, N*sizeof(int));
    hipMalloc((void **)&B_d, N*sizeof(int));
    hipMalloc((void **)&C_d, N*sizeof(int));

    // Copy data from cpu to gpu
    hipMemcpy(A_d, A, N*sizeof(int),hipMemcpyDefault);
    hipMemcpy(B_d, B, N*sizeof(int),hipMemcpyDefault);
    hipMemcpy(C_d, C, N*sizeof(int),hipMemcpyDefault);


    // Call Kernel method
    VecAdd<<<1, N>>>(A_d, B_d, C_d);

    // Copy data from gpu to cpu
    hipMemcpy(A, A_d, N*sizeof(int),hipMemcpyDefault);
    hipMemcpy(B, B_d, N*sizeof(int),hipMemcpyDefault);
    hipMemcpy(C, C_d, N*sizeof(int),hipMemcpyDefault);
    

    printVector(A, N);
    printVector(B, N);
    printVector(C, N);
    

    return 0;
}